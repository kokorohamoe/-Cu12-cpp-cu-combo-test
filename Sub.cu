
#include <hip/hip_runtime.h>
//Cu12 cpp cu combo test.cpp

#define threadPerBlock 2000
__global__ void kernel(int *dst,int *src,int N){
  int id = blockIdx.x * threadPerBlock * threadIdx.x;
  int x = src[id];
  int y;
  if(x >=0){
    y = 2*x*x*x+3*x*x*+x+1;
  }else{
    y= -x;
  }
  
  
}